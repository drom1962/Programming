#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "hip/hip_runtime.h"
#include ""

#include "nvcuvid.h"


#include <windows.h>

#include "MyCuda.h"

#include <stdio.h>


#include "..\..\container\ovi\Ovi_container.h"

typedef hipError_t  (CALLBACK*  cu_Init)(unsigned int Flags);
typedef hipError_t  (CALLBACK*  cu_CtxCreate)(hipCtx_t *pctx, unsigned int flags, hipDevice_t dev);


int main()
{
	int ret,ret1;

	// ������� ���������
    OVI2    *F1=new OVI2(1);
    FileInfo FI;
	
	ret=F1->Open(L"C:\\Video\\xx.ovi",&FI);
	if(ret!=0)
		{
		printf("File not opened\n");	
		return 0;
		}

	hipError_t result;


	
	// �������� ����������
	HINSTANCE		 dll=LoadLibrary("nvcuda.dll");
	if(dll==nullptr) return 0;

	// ������� ������� hipInit � �������� ��
	hipError_t resInit;
	cu_Init my_cuInit;
	my_cuInit=(cu_Init)GetProcAddress(dll,"hipInit");
	resInit=my_cuInit(0);
	if(resInit!=hipSuccess)  return 0;
	printf("CUDA Init success\n");
	
	cu_CtxCreate	my_cuCtxCreate;
	my_cuCtxCreate=(cu_CtxCreate)GetProcAddress(dll,"hipCtxCreate");

	hipCtx_t   	ctx;
	hipDevice_t  		dev=0;
	resInit=my_cuCtxCreate(&ctx,hipDeviceScheduleAuto,dev);
	if(resInit!=hipSuccess)  return 0;
	printf("CUDA Create context success\n");
	


	//// hipInit
	//result=hipInit(0);
	//if(result!=hipSuccess)  return 0;
	//printf("CUDA Init success\n");
	//
	//// �������� �������
	//hipCtx_t   	ctx;
	//hipDevice_t  		dev=0;
	//result=hipCtxCreate(&ctx,hipDeviceScheduleAuto,dev);
	//if(result!=hipSuccess)  return 0;
	//printf("CUDA Create context success\n");
	
	MyCuda *m1=new MyCuda();
    
    // ��������� I ����
    CUvideoparser           obj;
    CUVIDPARSERPARAMS       Parsparam;
    CUVIDSOURCEDATAPACKET   Sdp;

    VideoFrameInfo VFI;

    memset(&Parsparam,0,sizeof(CUVIDPARSERPARAMS));
    memset(&Sdp,0,sizeof(CUVIDSOURCEDATAPACKET));

    //   �������� CUVIDPARSERPARAMS
    //
    switch(FI.VideoCodec)
		{
		case H264:
				Parsparam.CodecType= cudaVideoCodec_H264;
				break;

		case MJPEG:
				Parsparam.CodecType= cudaVideoCodec_JPEG;
				break;

		default:
				return false;
		}

	// �������� ������� ��� ������� �����
	cuDecoderInfo DI;

    DI.Codec		=Parsparam.CodecType;
    DI.Width		=FI.Width;
    DI.Height		=FI.Height;

    DI.NewWidth		=FI.Width;
    DI.NewHeight	=FI.Height;
	DI.Flag			=VP;
	
	ret=m1->CreateDecoder(&DI);
	if(ret) 
		{
		return 0;
		}

    Parsparam.ulMaxNumDecodeSurfaces=1;
    Parsparam.ulMaxDisplayDelay=1;

	Parsparam.pfnSequenceCallback	=(PFNVIDSEQUENCECALLBACK)SequenceCallback;
	Parsparam.pfnDecodePicture		=(PFNVIDDECODECALLBACK)DecodePicture;
	Parsparam.pfnDisplayPicture		=(PFNVIDDISPLAYCALLBACK)DisplayPicture;

    result=cuvidCreateVideoParser(&obj,&Parsparam);

    F1->ReadVideoFrame(0,nullptr,0,&VFI);
    Sdp.flags           =CUVID_PKT_ENDOFSTREAM;
    Sdp.payload         =VFI.Data;
    Sdp.payload_size    =VFI.SizeFrame;

    result=cuvidParseVideoData(obj,&Sdp);
    
    result=cuvidDestroyVideoParser(&obj);
	
    
    void *Mdev1;
    ret=m1->AllocateMemory(&Mdev1,F1->GetMaxVideoFrame());
    
	cuVideoFrameInfo cuVFI;

    for(int i=0;i<FI.CountVideoFrame;i++)
        {
        ret=F1->ReadVideoFrame(i,nullptr,0,&VFI);

		cuVFI.Size=VFI.SizeFrame;
		cuVFI.Frame=VFI.Data;

		m1->DecoderFrame(DI.Decoder,cuVFI);


        ret1=m1->CopyMemoryHostToDevice(VFI.Data,Mdev1,VFI.SizeFrame);

		printf("%5d - %d  %d \n",i,ret,ret1);

        }

	getchar();

	return 0;
}